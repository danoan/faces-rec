#include "hip/hip_runtime.h"
#include "../headers/gpuBuffer.h"

GPUBuffer::GPUBuffer(int id, int nfeatures, int nimages):_id(id){
    _size = nfeatures*nimages;
    answer_host = (ulong*) malloc(sizeof(ulong)*_size);
    hipMalloc( (void**) &answer_device, sizeof(ulong)*_size );
}

GPUBuffer::~GPUBuffer(){
	Logger::cuda->log("*** \n FREE BUFFER \n *** \n");
	free(answer_host);
	hipFree(answer_device);
}
