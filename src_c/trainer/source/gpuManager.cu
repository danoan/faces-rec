#include "hip/hip_runtime.h"
#include "../headers/gpuManager.h"

typedef struct{
    int e1,e2;
} pair;

CUDA_CALLABLE_MEMBER_DEVICE pair pair_init(int e1, int e2){
    pair p;
    p.e1 = e1;
    p.e2 = e2;

    return p;
}

typedef struct{
    pair* v;

    int max_size;
    int read_cursor;
    int write_cursor;
    int size;
} queue;

CUDA_CALLABLE_MEMBER_DEVICE queue queue_init(int size){
    queue q;
    q.v = (pair*) malloc(sizeof(pair)*size);
    q.max_size = size;
    q.read_cursor = 0;
    q.write_cursor = 0;
    q.size=0;

    return q;
}

CUDA_CALLABLE_MEMBER_DEVICE void queue_push(queue* q, pair e){
    q->v[ q->write_cursor ] = e;
    q->write_cursor = (q->write_cursor+1)%q->max_size;
    q->size+=1;
}

CUDA_CALLABLE_MEMBER_DEVICE pair queue_pop(queue* q){
    pair p = q->v[q->read_cursor];
    q->read_cursor = (q->read_cursor+1)%q->max_size;
    q->size-=1;
    return p;
}

CUDA_CALLABLE_MEMBER_DEVICE void queue_destroy(queue* q){
	free(q->v);
}

template<class T>
CUDA_CALLABLE_MEMBER_DEVICE void merge(T* l, bool(* comp)(T* el1, T* el2), int b1, int e1, int b2, int e2){
    int size = e1-b1+1 + e2-b2+1;
    T* buffer = (T*) malloc(sizeof(T)*size);
    
    int cur_l = 0;
    int end_l = e1-b1+1;
    for(int i=cur_l;i<end_l;i++){
        buffer[i] = l[b1+i];
    }

    int cur_r = e1-b1+1;    
    int end_r = e2-b1+1;
    for(int i=cur_r;i<end_r;i++){
        buffer[i] = l[b1+i];
    }   
    
    int index=0;
    while( cur_l < end_l && cur_r < end_r ){
		
		if( comp( &buffer[cur_l], &buffer[cur_r] ) ){
            l[b1+index++] = buffer[cur_l++];
        }else{
            l[b1+index++] = buffer[cur_r++];
        }
    }

    while( cur_l < end_l ) l[b1+index++] = buffer[cur_l++];
    while( cur_r < end_r ) l[b1+index++] = buffer[cur_r++];
    
    free(buffer);
}

template<class T>
CUDA_CALLABLE_MEMBER_DEVICE void mergeSort(T* l, bool(* comp)(T* el1, T* el2), int begin, int end){
	int size = end-begin+1;
    
    queue q1 = queue_init(size+1);
    queue q2 = queue_init(size+1);
        
    for(int i=begin;i<=end;i++){
        queue_push(&q1, pair_init(i,i));
    }

    queue* q = &q1;
    queue* qb = &q2;

	while(1){
		
		if(q->size==1) break;
		
		while( q->size>0 ){			
			if(q->size==3){
				pair pl = queue_pop(q);
				pair pr = queue_pop(q);			
				pair plast = queue_pop(q);
				
				//printf("%d %d - %d %d\n",pl.e1,pl.e2, pr.e1, pr.e2);
				
				merge<T>(l, comp, pl.e1, pl.e2, pr.e1, pr.e2 );	
				pair presult = pair_init(pl.e1,pr.e2);				
				
				merge<T>(l, comp, presult.e1, presult.e2, plast.e1, plast.e2 );	
				queue_push(qb, pair_init(presult.e1,plast.e2));
			}else{
				
				pair pl = queue_pop(q);
				pair pr = queue_pop(q);
				
				//printf("%d %d - %d %d\n",pl.e1,pl.e2, pr.e1, pr.e2);

				merge<T>(l, comp, pl.e1, pl.e2, pr.e1, pr.e2 );

				queue_push(qb, pair_init(pl.e1,pr.e2));
			}			
		}
		
		queue* qt = q;
		q = qb;
		qb = qt;		
    }
    
    queue_destroy(&q1);
    queue_destroy(&q2);
    
    //pair p = queue_pop(q);
    //printf("%d %d - %d\n",p.e1,p.e2, q->size);
}

FeatureMaskDev convertFeatureMask(FeatureMask fm){
	FeatureMaskDev fmd;
	fmd._mask_size = fm._mask._size;
	fmd._location = fm._location;
	fmd._white_length = fm._mask._white.size();
	fmd._black_length = fm._mask._black.size();
	fmd._id = fm._id;
	
	for(int i=0;i<fmd._white_length;i++){
		fmd._white[i]._points[0] = fm._mask._white[i]._points[0];
		fmd._white[i]._points[1] = fm._mask._white[i]._points[1];
		fmd._white[i]._points[2] = fm._mask._white[i]._points[2];
		fmd._white[i]._points[3] = fm._mask._white[i]._points[3];
		
		fmd._white[i]._w = fm._mask._white[i]._w;
		fmd._white[i]._h = fm._mask._white[i]._h;
	}
	
	for(int i=0;i<fmd._black_length;i++){
		fmd._black[i]._points[0] = fm._mask._black[i]._points[0];
		fmd._black[i]._points[1] = fm._mask._black[i]._points[1];
		fmd._black[i]._points[2] = fm._mask._black[i]._points[2];
		fmd._black[i]._points[3] = fm._mask._black[i]._points[3];
		
		fmd._black[i]._w = fm._mask._black[i]._w;
		fmd._black[i]._h = fm._mask._black[i]._h;
	}	
	
	return fmd;
}

CUDA_CALLABLE_MEMBER_DEVICE long IntegralImageDev::getFromData(Point* points){
	long sum[4]={0,0,0,0};

	for(int i=0;i<4;++i){
		Point p = points[i];
		if(outsideLimits(p)){
			sum[i] = 0;
		}else{
			sum[i] = _data[p.y*24+p.x];
		}
	}

	return (sum[0]+sum[3]-sum[2]-sum[1]);	
}   

CUDA_CALLABLE_MEMBER_DEVICE long IntegralImageDev::computeBlock(MaskBlockDev* b, ulong mask_length, Point location){
	long block_sum = 0;

	for(int i=0;i<mask_length;++i){
		Point pa,pb,pc,pd;

		pa.y = location.y + b[i]._points[0].y - 1;
		pa.x = location.x + b[i]._points[0].x - 1;

		pb.y = location.y + b[i]._points[1].y - 1;
		pb.x = location.x + b[i]._points[1].x;

		pc.y = location.y + b[i]._points[2].y;
		pc.x = location.x + b[i]._points[2].x - 1;

		pd.y = location.y + b[i]._points[3].y;
		pd.x = location.x + b[i]._points[3].x;						


		Point points[4] = {pa,pb,pc,pd};

		// printf("%ld (%lu %lu) [ (%lu %lu) (%lu %lu) (%lu %lu) (%lu %lu) ]\n",getFromData(points),location.x,location.y,pa.y,pa.x,pb.y,pb.x,pc.y,pc.x,pd.y,pd.x);
		// printf("%ld (%lu %lu)",b[i]._points[0].y,b[i]._points[0].x);
		
		block_sum+=getFromData(points);		
	}

	return block_sum;	
}

CUDA_CALLABLE_MEMBER_DEVICE ulong IntegralImageDev::filter(FeatureMaskDev* fm){
	if( (fm->_mask_size.x > _size.x) || (fm->_mask_size.y > _size.y) ){
		return 120;
	}

	// printf("%d\n", abs( computeBlock(fm._mask._black,fm._location)  - computeBlock(fm._mask._white,fm._location) ) );

	return abs( computeBlock(fm->_black,fm->_black_length,fm->_location)  - computeBlock(fm->_white,fm->_white_length,fm->_location) );	
}

void loadImage(ulong* data, const char* filepath,Point* size){
	MagickWand * image_wand;
	PixelIterator* iterator;
	MagickBooleanType status;
	MagickPixelPacket pixel;
	PixelWand** pixels;

	long x,y;
	ulong line_sum;

	image_wand = NewMagickWand();
	status=MagickReadImage(image_wand,filepath);	

	iterator=NewPixelIterator(image_wand);

	size->y = MagickGetImageHeight(image_wand);
	size->x = MagickGetImageWidth(image_wand);

	// printf("MAGICK %lu %lu\n",size->y,size->x);

	if(data==NULL){
		return;
	}

	for (y=0; y < size->y; ++y){		
	    pixels=PixelGetNextIteratorRow(iterator,&(size->x));

	    if ( (pixels == (PixelWand **) NULL) ) break;

	    line_sum=0;
	    for (x=0; x < (long) size->x; ++x){
	    	line_sum+= PixelGetBlue(pixels[x])*255;

	    	if(y>0){
	    		data[y*size->x+x] = data[(y-1)*size->x+x] + line_sum;
	    	}else{
	    		data[y*size->x+x] = line_sum;
	    	}
	
	    	// printf("%lu\n",(*data)[y][x]);
    	}	
    }
    DestroyPixelIterator(iterator);
    DestroyMagickWand(image_wand);	    
}

void prepareData(TrainingSet& ts, Point* size_image, ulong** data_host, ulong** data_device, int nfeatures, FeatureMaskDev** fmd_host, FeatureMaskDev** fmd_device, bool alloc){
	int ardis_w = 24;
	int ardis_h = 24;
	int nimages = ts.size();
	int totalPixels = nimages*ardis_w*ardis_h;
	
	printf("PREPARING DATA\n");
	
	//Preparing Images	
	if(alloc){
		(*data_host) = (ulong*) malloc(sizeof(ulong)*totalPixels);
		hipMalloc( (void**) data_device, sizeof(ulong)*totalPixels );	
	}
	
	
	for(int i=0;i<nimages;i++){		
		TrainingImage* ti = ts.get(i);
		for(int h=0;h<ardis_h;h++){
			for(int w=0;w<ardis_w;w++){	
				( (*data_host)+(i*ardis_w*ardis_h) )[h*ardis_w+w] = ti->_ii->_data[h][w];
			}
		}
	}	
	/*
	for(int i=0;i<totalPixels;i++){
		printf("%d\n", (*data_host)[i] );
	}
	*/	
	
	hipMemcpy( *data_device, *data_host, sizeof(ulong)*totalPixels, hipMemcpyHostToDevice );	
	
		
	//Preparing Features
	FacesFeatureFactory fff;
	
	if(alloc) (*fmd_host) = (FeatureMaskDev*) malloc(sizeof(FeatureMaskDev)*nfeatures);
	
	for(int i=0;i<nfeatures;i++){
		(*fmd_host)[i] = convertFeatureMask(fff._facesFeatures[i]);
	}
	
	if(alloc) hipMalloc( (void**) fmd_device, sizeof(FeatureMaskDev)*nfeatures );
	
	hipMemcpy( (*fmd_device), (*fmd_host), sizeof(FeatureMaskDev)*nfeatures, hipMemcpyHostToDevice );
}

__global__ void kernelFilter(ulong* data, Point size, int nimages, FeatureMaskDev* fmd,int startFeature, int stepFeature, int nfeatures, ulong* answer){	

	/*
	int factorF = stepFeature/712 + 1;
	int featDe = threadIdx.x*factorF;
	int featAte = (threadIdx.x+1)*factorF;
	
	int factorI = nimages/16000 + 1;
	int imgDe =  blockIdx.x*factorI;
	int imgAte = (blockIdx.x+1)*factorI;
	*/
	
	
	int factorF = stepFeature/gridDim.x + 1;
	int featDe = blockIdx.x*factorF;
	int featAte = (blockIdx.x+1)*factorF;
	
	int factorI = nimages/blockDim.x + 1;
	int imgDe =  threadIdx.x*factorI;
	int imgAte = (threadIdx.x+1)*factorI;	
	
	if(featDe < stepFeature && imgDe < nimages){
		
		if(featAte>stepFeature) featAte = stepFeature;
		if(imgAte>nimages) imgAte = nimages;
		
		for(int featId=featDe;featId<featAte;featId++){
			if( (featId+startFeature)>=nfeatures) break;
			for(int imgId=imgDe;imgId<imgAte;imgId++){
				IntegralImageDev ii( &(data[24*24*imgId]) ,size);
				
				answer[featId*nimages+imgId] = ii.filter( &(fmd[featId+startFeature]) );	
				//answer[threadIdx.x] = ii.filter( &(fmd[0]) );	
				
			}			
		}
	}
	
	/*
	answer[0] = data[0];
	answer[1] = data[1];
	answer[2] = data[2];
	answer[3] = data[3];
	*/
}



int callCUDA(ulong* data_device, Point size_image, int nimages, FeatureMaskDev* fmd_device, int startFeature, int stepFeature, int nfeatures, int sizeAnswer, ulong* answer_host, ulong* answer_device){	
	Logger::cuda->log("KERNEL CALL %d %d %d %d %d %d %d\n",size_image.x,size_image.y,nimages,startFeature,stepFeature,nfeatures,sizeAnswer);
	kernelFilter<<<GRID_SIZE,BLOCK_SIZE>>>(data_device, size_image, nimages, fmd_device, startFeature, stepFeature, nfeatures, answer_device);
	hipMemcpy( answer_host, answer_device, sizeof(ulong)*sizeAnswer, hipMemcpyDeviceToHost );			
	Logger::cuda->log("END CALL\n");
	
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}	
}


void* gpuThread(void* vp){
    GPUManager* manager = (GPUManager*) vp;

	Logger::cuda->log("BEGINBEGUNBEGIN\n");
    while(manager->END_STAGE==0){
        if(manager->CONSUMED_BUFFER==1){
			Logger::cuda->log("BUFFER LOAD PROCESS BEGIN\n");
            pthread_mutex_lock( &(manager->M) );
            Logger::cuda->log("INSIDE LOCK\n");
            int from,to;
            GPUBuffer* buffer = manager->getConsumedBuffer(&from,&to);
            pthread_mutex_unlock( &(manager->M) );
            manager->fillBuffer(buffer,from,to);
            Logger::cuda->log("BUFFER LOAD PROCESS END\n");
        }
    }
	Logger::cuda->log("FIMFIMFIM\n");
    pthread_exit(NULL);

}

void GPUManager::wait(){
	pthread_join( threads[0], NULL );
}

GPUManager::GPUManager(TrainingSet& ts, int totalFeatures, int nimages):_total_features(totalFeatures),_nimages(nimages){	  
    _feat_per_buffer = SINGLE_BUFFER_STEP_FEATURE/GPU_BUFFER + 1;
    _max_stage = totalFeatures/_feat_per_buffer + 1;
 
 	size_image.x = 24;
	size_image.y = 24;    printf("GPU BUFFER %d\n",Config::CUDA_BUFFER);
	buffers = (GPUBuffer**) malloc(sizeof(GPUBuffer*)*GPU_BUFFER);
    for(int i=0;i<GPU_BUFFER;i++){
        buffers[i] = new GPUBuffer(i,_feat_per_buffer, nimages);
    }
    resetManager();
    	
    prepareData(ts, &size_image, &data_host, &data_device, totalFeatures, &fmd_host, &fmd_device, true);
}

void GPUManager::resetManager(){
	_cur_stage = 0; 
	
    _buffer_chunk = 0;
    _get_filled_counter = 0;	
    
    while(!consumed_buffers.empty()) consumed_buffers.pop();
    while(!filled_buffers.empty()) filled_buffers.pop();
    
    for(int i=0;i<GPU_BUFFER;i++){
        consumed_buffers.push(buffers[i]);
    }    
    
    CONSUMED_BUFFER = true;
    END_STAGE=false;    
}

void GPUManager::resetImageData(TrainingSet& ts){
	_nimages = ts.size();
	Logger::cuda->log("NEW SIZE %d\n",_nimages);
    
	free(data_host);
	free(fmd_host);
	
	hipFree(data_device);
	hipFree(fmd_device);
	
    for(int i=0;i<GPU_BUFFER;i++){
		Logger::cuda->log("REMOVING BUFFER\n");
		delete buffers[i];
        buffers[i] = new GPUBuffer(i,_feat_per_buffer, _nimages);
    }	
    resetManager();
	
	prepareData(ts, &size_image, &data_host, &data_device, _total_features, &fmd_host, &fmd_device, true);	
}

void GPUManager::restart(){
    Logger::cuda->log("INITIALIZING GPU MANAGER...START THREAD\n");
	pthread_create(&(threads[0]),NULL, gpuThread, this);
}

GPUBuffer* GPUManager::getConsumedBuffer(int* from, int* to){
	
    if(consumed_buffers.size()==0) return NULL;
    if(consumed_buffers.size()==1){ CONSUMED_BUFFER = false; Logger::cuda->log("CONSUMED FALSE %d\n", CONSUMED_BUFFER); }

    GPUBuffer* b = consumed_buffers.front();
    consumed_buffers.pop();

    *from = _cur_stage*SINGLE_BUFFER_STEP_FEATURE + b->_id*_feat_per_buffer;
    *to = _cur_stage*SINGLE_BUFFER_STEP_FEATURE + (b->_id+1)*_feat_per_buffer;

	if(_buffer_chunk == GPU_BUFFER-1){
		_cur_stage = _cur_stage+1;
	}
	
	_buffer_chunk = (_buffer_chunk+1)%GPU_BUFFER;

    if(*to>_total_features) *to=_total_features;

    b->_from = *from;
    b->_to = *to;

    return b;
}

void GPUManager::bufferHasBeenConsumed(GPUBuffer* b){
	consumed_buffers.push(b);
	pthread_mutex_lock( &(M) );
	CONSUMED_BUFFER = true;
	Logger::cuda->log("CONSUMED TRUE %d\n", CONSUMED_BUFFER);
	pthread_mutex_unlock( &(M) );
}

void GPUManager::fillBuffer(GPUBuffer* b, int from, int to){
	Logger::cuda->log("CALL CUDA BUFFER ID: %d\n",b->_id);
	if(from<_total_features){
		callCUDA(data_device, size_image, _nimages, fmd_device, from, _feat_per_buffer, _total_features, b->_size, b->answer_host, b->answer_device);
	}
    filled_buffers.push(b);
}

GPUBuffer* GPUManager::getFilledBuffer(){
    if(filled_buffers.size()==0) return NULL;

	Logger::cuda->log("%d - %d\n",_get_filled_counter,_max_stage);

    if( _get_filled_counter==(_max_stage-1) ) END_STAGE = true;

    _get_filled_counter= (_get_filled_counter+1)%(_max_stage);

    GPUBuffer* b = filled_buffers.front();
    filled_buffers.pop();

    return b;   
}
