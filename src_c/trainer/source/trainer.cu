#include "hip/hip_runtime.h"
#include "../headers/trainer.h"


void Trainer::inputInfo(){
    Logger::logger->log("INPUT INFO\n\n");
    Logger::logger->log("ARDIS_WIDTH: %d\nARDIS_HEIGHT: %d\n",_ardis.x,_ardis.y);
    Logger::logger->log("SHIFT_STEP: %d\nRESIZE_FACTOR: %f\nSTART_WIDTH: %d\nSTART_HEIGHT: %d\n",_shift_w,_resize_factor,_start_w,_start_h);
    Logger::logger->log("FINAL_FALSE_POSITIVE_RATE: %f\nFINAL_DETECTION_RATE: %f\n",_final_fp_rate,_final_det_rate);
    Logger::logger->log("STAGE_MAX_FALSE_POSITIVE_RATE: %f\nSTAGE_MIN_DETECTION_RATE: %f\n",_max_fp_rate,_min_det_rate);
    Logger::logger->log("MAX_STATES: %d\nBETA_MIN_VALUE: %f\n\n",Config::CLASSIFIER_MAX_STAGES, Config::CLASSIFIER_BETA_MIN_VALUE);

    Logger::logger->log("FEATURES INFO\n\n");

    Logger::logger->log("Total Features: %d\n",_facesFactory._facesFeatures.size());    
}

Trainer::Trainer(TrainerSetManager* tsm){
    _ardis.x = Config::ARDIS_WIDTH; _ardis.y = Config::ARDIS_HEIGHT;

    _shift_w = Config::CLASSIFIER_SHIFT_STEP;
    _shift_h = Config::CLASSIFIER_SHIFT_STEP;

    _resize_factor = Config::CLASSIFIER_RESIZE_FACTOR;
    _start_w = Config::CLASSIFIER_SUBWINDOW_START_WIDTH;
    _start_h = Config::CLASSIFIER_SUBWINDOW_START_HEIGHT;    

    _final_fp_rate = Config::CLASSIFIER_FINAL_FALSE_POSITIVE_RATE;
    _final_det_rate = Config::CLASSIFIER_FINAL_DETECTION_RATE;

    _max_fp_rate = Config::CLASSIFIER_STAGE_MAX_FALSE_POSITIVE_RATE;
    _min_det_rate = Config::CLASSIFIER_STAGE_MIN_DETECTION_RATE;    

    _tsm = tsm;
    _tsm->init(_facesFactory._facesFeatures.size(),Config::CLASSIFIER_INTEGRAL_IMAGE_BUFFER_SIZE);
    
    _stage_number = 0;
    _feature_number = 0;

    inputInfo();
    
	_firstStagesMaxFeature[0] = 2;
	_firstStagesMaxFeature[1] = 5;
	_firstStagesMaxFeature[2] = 10;
	_firstStagesMaxFeature[3] = 25;
	_firstStagesMaxFeature[4] = 50;
	
	gpuManager = new GPUManager( _tsm->trs(), _facesFactory._facesFeatures.size(), tsm->trs().size() );
	cpuManager = new CPUManager();
}

void Trainer::prepareTrainer(){
    for(int i=0;i<CPU_THREADS;i++) _ct[i] = new ClassificationTable();        
    for(int i=0;i<CPU_THREADS;i++) _ct[i]->initTable(_tsm->trs());
}

Classifier Trainer::startTraining(){
    prepareTrainer();
    Logger::debug->log("Start Training Stage %d\n\n", _stage_number++);
    _feature_number=0;

    Classifier fc;   
    keepTraining(fc);

    return fc;
}

CascadeClassifier Trainer::startTrainingCascade(){    
    CascadeClassifier cascade;
    double _fp_rate = 1.0;
    double _det_rate = 1.0;

    double ac;
    double fi;
    double di;

    int total_features = 0;
    int r;
    while( _fp_rate>_final_fp_rate && _stage_number < Config::CLASSIFIER_MAX_STAGES ){          
        prepareTrainer(); 

        Logger::debug->log("Start Training Stage %d\n\n", _stage_number);
        _feature_number=0;

        Classifier fc;    
        startClock();       
        keepTraining(fc);
        stopClock("KEEP TRAINING");

        if(_stage_number<5){
            while(!firstStagesCheckClassifier(fc,&ac,&fi,&di,_stage_number,_feature_number)){
                startClock();       
                keepTraining(fc);
                stopClock("KEEP TRAINING");
            }
        }else{
            while(!checkClassifier(fc,&ac,&fi,&di) && _feature_number < Config::CLASSIFIER_MAX_HYPOTHESIS_PER_STAGE){
                startClock();       
                keepTraining(fc);
                stopClock("KEEP TRAINING");
            }            
        }


        _fp_rate*=fi;
        _det_rate*=di;

        Logger::debug->log("FINAL F: %.4f\nFINAL D: %.4f\n",_fp_rate,_det_rate);

        fc._ac = ac;
        cascade.addClassifier(fc);

        char path[128];
        total_features += _feature_number;
        sprintf(path,"%s/classifier_%d_%d_%d",Config::STATES_PATH.c_str(), _tsm->trs()._faces.size()+_tsm->vas()._scenes.size(), _stage_number, total_features);
        printf("%s\n",path);
        cascade.save(std::string(path));

        _stage_number++;    
        r = _tsm->resetSets(_stage_number,cascade);
        printf("END RESET\n");
        
        endTrainer();    
        gpuManager->resetImageData(_tsm->trs());
        if(r==-1) break;   
    }
    

    return cascade;
}


void Trainer::keepTraining(Classifier& cl){
    Logger::debug->log("FEATURE %d\n\n", _feature_number);

    bool isAllFeaturesCovered = false;    
    TableItem theBest = TableItem(0,0,FACE,1.0,1);
    gpuManager->restart();

    while(!gpuManager->END_STAGE){
        GPUBuffer* gpb = gpuManager->getFilledBuffer();    

        if(gpb!=NULL){
			printf("CONSUME BUFFER\n");
			TableItem partial = cpuManager->consumeBuffer(gpb,gpuManager,this);
			if(partial._error < theBest._error) theBest = partial;
		}
    }
    
    gpuManager->wait();
    
    double e_t = theBest._error;
    double b_t = (e_t)/(1-e_t);
    double a_t = 0;

    if (b_t <= Config::CLASSIFIER_BETA_MIN_VALUE){
        a_t = log( 1.0/(Config::CLASSIFIER_BETA_MIN_VALUE) );
    }else{
        a_t = log( 1.0/b_t );
    }
    Logger::debug->log( "IT: %d-%d \n E_T: %.12f \n B_T: %.12f \n A_T %.12f \n\n",_stage_number,_feature_number,e_t,b_t,a_t);

    Hypothesy h (theBest._filter_value, theBest._direction, a_t, theBest._fm );
    Logger::debug->log("HYPOTHESY %lu %d %lf %d \n",h._threshold,h._direction,h._alpha,h._fm._id);

    cl.addHypothesy(h);
    cl._ardis = _ardis;

    for(int i=0;i<CPU_THREADS;i++) _ct[i]->updateWeights(b_t,h,_tsm->trs());
    gpuManager->resetManager();
    
    _feature_number+=1;
}

bool Trainer::firstStagesCheckClassifier(Classifier& cc, double* ac, double* fi, double* di, int stage, int featureNumber){
    bool check = _tsm->checkClassifier(cc,ac,fi,di,0.75,_min_det_rate,0.4,0.025,0.1);
    if(check==false){
         if( featureNumber >= _firstStagesMaxFeature[stage] ) return true;
    }

    return check;
}

bool Trainer::checkClassifier(Classifier& cc, double* ac, double* fi, double* di){
    return _tsm->checkClassifier(cc,ac,fi,di,_max_fp_rate,_min_det_rate,0.525,0.025,0.25);
}
