#include "hip/hip_runtime.h"
#include "../headers/cpuManager.h"

void* cpuThread(void* vp){
    elem_params* ep = (elem_params*) vp;

	Logger::cuda->log("GO %d - %d\n", ep->from, ep->to);

    TableItem partialBest;
    for(int i=ep->from;i<ep->to;i++){
        //printf("GO %d\n", i);
        partialBest = ep->t->_ct[ep->thread_number]->getBestTableItem( (ep->t->_facesFactory._facesFeatures)[i], ep->t->_tsm->trs(), ep->answer_host, ep->startFeature );
        //printf("STOP\n");

        if(partialBest._error<ep->best._error){
            ep->best = partialBest;
        }   
    }

    Logger::cuda->log("THE END\n");

    pthread_exit(NULL);

}

void CPUManager::createThreads(GPUBuffer* b, Trainer* t, elem_params* ep){
    int length = b->_to - b->_from + 1;
    int factor = length/CPU_THREADS + 1;

    
    for(int i=0;i<CPU_THREADS;i++){
        
        ep[i].thread_number = i;
        ep[i].best = TableItem(0,0,FACE,1.0,1);
        ep[i].t = t;
        ep[i].answer_host = b->answer_host;
        ep[i].from = b->_from + i*factor;
        ep[i].to = b->_from + (i+1)*factor;
        ep[i].startFeature = b->_from;
        
        if(ep[i].to>b->_to) ep[i].to=b->_to;

        pthread_create(&(_threads[i]), NULL, cpuThread, &ep[i]);
    }
}

TableItem CPUManager::consumeBuffer(GPUBuffer* b, GPUManager* gpum, Trainer* t){
	elem_params ep[CPU_THREADS];
    createThreads(b,t,ep);
    for(int i=0;i<CPU_THREADS;i++){
        pthread_join(_threads[i],NULL);
    }
    
    Logger::cuda->log("END JOIN\n");
    
    gpum->bufferHasBeenConsumed(b);

    TableItem theBest = ep[0].best;
    for(register int t=1;t<CPU_THREADS;t++){
        if(ep[t].best._error < theBest._error){
            theBest = ep[t].best;
        }
    }    

    return theBest;
}
